#include <hip/hip_runtime.h>

#include <iostream>

__device__ int cuda_mandelbrot_iterations(double real, double imag, int max_iterations) {
    double z_real = 0.0, z_imag = 0.0;
    int iterations = 0;
    
    while (z_real * z_real + z_imag * z_imag <= 4.0 && iterations < max_iterations) {
        double temp = z_real * z_real - z_imag * z_imag + real;
        z_imag = 2.0 * z_real * z_imag + imag;
        z_real = temp;
        iterations++;
    }
    
    return iterations;
}

__device__ int cuda_julia_iterations(double z_real, double z_imag, double c_real, double c_imag, int max_iterations) {
    int iterations = 0;
    
    while (z_real * z_real + z_imag * z_imag <= 4.0 && iterations < max_iterations) {
        double temp = z_real * z_real - z_imag * z_imag + c_real;
        z_imag = 2.0 * z_real * z_imag + c_imag;
        z_real = temp;
        iterations++;
    }
    
    return iterations;
}

__device__ void cuda_iterations_to_color(int iterations, int max_iterations, 
                                          unsigned char* r, unsigned char* g, unsigned char* b) {
    if (iterations == max_iterations) {
        *r = *g = *b = 0;
        return;
    }
    
    double t = (double)iterations / max_iterations;
    *r = (unsigned char)(9 * (1 - t) * t * t * t * 255);
    *g = (unsigned char)(15 * (1 - t) * (1 - t) * t * t * 255);
    *b = (unsigned char)(8.5 * (1 - t) * (1 - t) * (1 - t) * t * 255);
}

__global__ void mandelbrot_cuda_kernel(unsigned char* image, int width, int height, int max_iterations,
                                       double x_min, double x_max, double y_min, double y_max) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    double real = x_min + (x_max - x_min) * x / (width - 1);
    double imag = y_min + (y_max - y_min) * y / (height - 1);
    
    int iterations = cuda_mandelbrot_iterations(real, imag, max_iterations);
    
    unsigned char r, g, b;
    cuda_iterations_to_color(iterations, max_iterations, &r, &g, &b);
    
    int idx = (y * width + x) * 3;
    image[idx] = r;
    image[idx + 1] = g;
    image[idx + 2] = b;
}

__global__ void julia_cuda_kernel(unsigned char* image, int width, int height, int max_iterations,
                                  double x_min, double x_max, double y_min, double y_max,
                                  double c_real, double c_imag) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    double z_real = x_min + (x_max - x_min) * x / (width - 1);
    double z_imag = y_min + (y_max - y_min) * y / (height - 1);
    
    int iterations = cuda_julia_iterations(z_real, z_imag, c_real, c_imag, max_iterations);
    
    unsigned char r, g, b;
    cuda_iterations_to_color(iterations, max_iterations, &r, &g, &b);
    
    int idx = (y * width + x) * 3;
    image[idx] = r;
    image[idx + 1] = g;
    image[idx + 2] = b;
}

extern "C" void launch_mandelbrot_kernel(unsigned char* d_image, int width, int height, int max_iterations,
                                         double x_min, double x_max, double y_min, double y_max) {
    int block_size_x = 16;
    int block_size_y = 16;
    
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    
    if (prop.major >= 8) {
        block_size_x = 16;
        block_size_y = 16;
    }
    
    dim3 block_size(block_size_x, block_size_y);
    dim3 grid_size((width + block_size.x - 1) / block_size.x, 
                   (height + block_size.y - 1) / block_size.y);
    
    mandelbrot_cuda_kernel<<<grid_size, block_size>>>(d_image, width, height, max_iterations, 
                                                      x_min, x_max, y_min, y_max);
    
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA Mandelbrot kernel failed: " << hipGetErrorString(err) << std::endl;
    }
}

extern "C" void launch_julia_kernel(unsigned char* d_image, int width, int height, int max_iterations,
                                    double x_min, double x_max, double y_min, double y_max,
                                    double c_real, double c_imag) {
    int block_size_x = 16;
    int block_size_y = 16;
    
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    
    if (prop.major >= 8) {
        block_size_x = 16;
        block_size_y = 16;
    }
    
    dim3 block_size(block_size_x, block_size_y);
    dim3 grid_size((width + block_size.x - 1) / block_size.x, 
                   (height + block_size.y - 1) / block_size.y);
    
    julia_cuda_kernel<<<grid_size, block_size>>>(d_image, width, height, max_iterations, 
                                                 x_min, x_max, y_min, y_max, c_real, c_imag);
    
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA Julia kernel failed: " << hipGetErrorString(err) << std::endl;
    }
}